#include "hip/hip_runtime.h"
///////////////////////////////////////////////////////////////////////////////
//
// CUDA function definitions
//
///////////////////////////////////////////////////////////////////////////////
#include "kernel.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
/**
template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, unsigned int n) {
extern __shared__ int sdata[];
unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*(blockSize * 2) + tid;
unsigned int gridSize = blockSize * 2 * gridDim.x;
sdata[tid] = 0;
while (i < n) { sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSize; }
__syncthreads();
if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
if (tid < 32) warpReduce(sdata, tid);
if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
*/
///////////////////////////////////////////////////////////////////////////////
// error check function
#define cuda_safe_call(ans) { simCudaAssert((ans), __FILE__, __LINE__); }
void simCudaAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

///////////////////////////////////////////////////////////////////////////////
// used namespace
using namespace simula;
using namespace simCuda;

///////////////////////////////////////////////////////////////////////////////
// local namespace
namespace {
	typedef MoleculeType::core_t kMoleculeType;
	kMoleculeType *tlist_d, *tlist_h;
	void copy_molecule_type()
	{
		simSize msize = molecules.type_num();
		tlist_h = (kMoleculeType*)malloc(msize * sizeof(kMoleculeType));
		for (simSize i = 0; i < msize; ++i) {
			tlist_h[i] = molecules.type(i).core();
			// deep copy dot pos
			simI3* dot_pos_d;
			simSize len = tlist_h[i].dot_num;
			cuda_safe_call(hipMalloc((simI3**)&dot_pos_d, len * sizeof(simI3)));
			cuda_safe_call(hipMemcpy(dot_pos_d, tlist_h[i].dot_pos, len * sizeof(simI3), hipMemcpyHostToDevice));
			tlist_h[i].dot_pos = dot_pos_d;
		}
		cuda_safe_call(hipMalloc((kMoleculeType**)&tlist_d, msize * sizeof(kMoleculeType)));
		cuda_safe_call(hipMemcpy(tlist_d, tlist_h, msize * sizeof(kMoleculeType), hipMemcpyHostToDevice));
	}
	/////////////////////////////////////////////////////////////////////////////
	// device variable
	kMolecule* mlist_d;
	kMolecule* mlist_h;

	/////////////////////////////////////////////////////////////////////////////
	// deep copy molecule list to device
	hipError_t mlist_to_dev(simBool free_flag = false)
	{
		// allocate memory
		simSize msize = molecules.molecule_num();
		mlist_h = (kMolecule*)malloc(msize * sizeof(kMolecule));
		cuda_safe_call(hipMalloc((void**)&mlist_d, msize * sizeof(kMolecule)));
		// deep copy molecule data into C struct
		for (simI1 i = 0; i < msize; ++i) {
			mlist_h[i].x = molecules.molecule(i).x();
			mlist_h[i].y = molecules.molecule(i).y();
			mlist_h[i].d = molecules.molecule(i).d();
			mlist_h[i].i = molecules.molecule(i).self_id();
			mlist_h[i].t = molecules.molecule(i).type_id();
		}
		// copy data into device
		hipError_t err = hipMemcpy(mlist_d, mlist_h, msize * sizeof(kMolecule), hipMemcpyHostToDevice);
		// free mlist_h
		if (free_flag) {
			free(mlist_h);
		}
		return err;
	}

	/////////////////////////////////////////////////////////////////////////////
	// deep copy molecule list to host
	hipError_t mlist_to_host()
	{
		// allocate memory
		simSize msize = molecules.molecule_num();
		// copy data back to host
		hipError_t err = hipMemcpy(mlist_h, mlist_d, msize * sizeof(kMolecule), hipMemcpyDeviceToHost);
		// deep copy back to struct
		if (err = hipSuccess) {
			for (simI1 i = 0; i < msize; ++i) {
				molecules.molecule(i).set_x(mlist_h[i].x);
				molecules.molecule(i).set_y(mlist_h[i].y);
				molecules.molecule(i).set_d(mlist_h[i].d);
			}
		}
		return err;
	}

	/////////////////////////////////////////////////////////////////////////////
	// kernel function
	__device__ __host__ simI1 pmod_k(simI1 x, simI1 n) {
		return ((x % n) + n) % n;
	}
	// ==> to check if its neighboring points are occupied
	__device__ simBool check_empty(kMoleculeType* tlist, simI1 sx, simI1 sy, kMolecule& ms, kMolecule& mt, simI1 xlen, simI1 ylen)
	{
		simBool empty = true;
		kMoleculeType& type_s = tlist[ms.t - 1];
		kMoleculeType& type_t = tlist[mt.t - 1];
		simSize len_s = type_s.dot_num;
		simSize len_t = type_t.dot_num;

		for (simI1 i = 0; i < len_s; ++i) {
			simI1 x_s, y_s;
			x_s = pmod_k(sx + type_s.dot_pos[i].x,xlen);
			y_s = pmod_k(sy + type_s.dot_pos[i].y,ylen);
			for (simI1 j = 0; j < len_t; ++j) {
				simI1 x_t, y_t;
				x_t = pmod_k(mt.x + type_t.dot_pos[j].x, xlen);
				y_t = pmod_k(mt.y + type_t.dot_pos[j].y, ylen);
				if (x_s == x_t && y_s == y_t) { empty = false; break; }
			}
		}

		return empty;
	}
	__global__ void addKernel(kMoleculeType* tlist, kMolecule* mlist, simI1* r, simI1 size, simI1 xsize, simI1 ysize)
	{
		simI1 idx = threadIdx.x;
		simI1 sx = mlist[idx].x, sy = mlist[idx].y;

		r[idx * 4 + 0] = idx+1;
		r[idx * 4 + 1] = idx+1;
		r[idx * 4 + 2] = idx+1;
		r[idx * 4 + 3] = idx+1;

		for (simI1 i = 0; i < size; ++i) {
			if (idx != i) {
				if (!check_empty(tlist, sx + 1, sy, mlist[idx], mlist[i], xsize, ysize)) { r[idx * 4 + 0] = 0; }
				if (!check_empty(tlist, sx, sy + 1, mlist[idx], mlist[i], xsize, ysize)) { r[idx * 4 + 1] = 0; }
				if (!check_empty(tlist, sx - 1, sy, mlist[idx], mlist[i], xsize, ysize)) { r[idx * 4 + 2] = 0; }
				if (!check_empty(tlist, sx, sy - 1, mlist[idx], mlist[i], xsize, ysize)) { r[idx * 4 + 3] = 0; }
			}
		}

	}

	simI1* result_d;
	simI1* result_h;

	// Helper function for using CUDA to add vectors in parallel.
	void funcCuda()
	{
		simI1 msize = molecules.molecule_num();

		// Choose which GPU to run on, change this on a multi-GPU system.
		cuda_safe_call(hipSetDevice(0));

		// Allocate constant memory
		result_h = (simI1*)malloc(4 * msize * sizeof(simI1));
		cuda_safe_call(hipMalloc((void**)&result_d, 4 * msize * sizeof(simI1)));

		// Copy data
		cuda_safe_call(mlist_to_dev(true));

		copy_molecule_type();

		// Launch a kernel on the GPU with one thread for each element.
		addKernel <<< 1, msize >>> (tlist_d, mlist_d, result_d, msize, sub.xlen(), sub.ylen());

		// Check for any errors launching the kernel
		cuda_safe_call(hipGetLastError());

		// Check any errors encountered during the launch.
		cuda_safe_call(hipDeviceSynchronize());

		// Copy output vector from GPU buffer to host memory.
		cuda_safe_call(hipMemcpy(result_h, result_d, 4 * msize * sizeof(simI1), hipMemcpyDeviceToHost));
	}

};

int simCuda::main_temp()
{
	// Test overlap.
	funcCuda();
	// print result
	for (simI1 i = 0; i < molecules.molecule_num(); ++i) {
		printf("{%d,%d,%d,%d}\n", result_h[4 * i + 0], result_h[4 * i + 1], result_h[4 * i + 2], result_h[4 * i + 3]);
	}

	// for tracing tools such as Nsight and Visual Profiler
	cuda_safe_call(hipDeviceReset());
	return 0;
}